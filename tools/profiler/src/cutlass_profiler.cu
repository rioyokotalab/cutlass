/***************************************************************************************************
 * Copyright (c) 2017 - 2024 NVIDIA CORPORATION & AFFILIATES. All rights reserved.
 * SPDX-License-Identifier: BSD-3-Clause
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *
 * 1. Redistributions of source code must retain the above copyright notice, this
 * list of conditions and the following disclaimer.
 *
 * 2. Redistributions in binary form must reproduce the above copyright notice,
 * this list of conditions and the following disclaimer in the documentation
 * and/or other materials provided with the distribution.
 *
 * 3. Neither the name of the copyright holder nor the names of its
 * contributors may be used to endorse or promote products derived from
 * this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
 * AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
 * DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE
 * FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL
 * DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR
 * SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER
 * CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY,
 * OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 *
 **************************************************************************************************/
/* \file
   \brief Execution environment
*/

#include <iostream>
#include <stdexcept>

// Profiler includes
#include "cutlass/profiler/cutlass_profiler.h"
#include "cutlass/profiler/gemm_operation_profiler.h"

/////////////////////////////////////////////////////////////////////////////////////////////////

namespace cutlass {
namespace profiler {

/////////////////////////////////////////////////////////////////////////////////////////////////

CutlassProfiler::CutlassProfiler(
  Options const &options
):
  options_(options) {

  operation_profilers_.emplace_back(new GemmOperationProfiler(options));




}

CutlassProfiler::~CutlassProfiler() {

}

/////////////////////////////////////////////////////////////////////////////////////////////////

/// Execute the program
int CutlassProfiler::operator()() {

/*  if (options_.cmdline.num_naked_args() > 0) {
    std::cerr << "Unknown args: \n";
    options_.cmdline.print_naked_args(std::cerr);
    std::cerr << "\n\n\n";

   print_usage_(std::cout);
    return 1;
  }
*/
  // if (options_.about.help) {
  //   if (options_.operation_kind == library::OperationKind::kInvalid) {
  //     print_usage_(std::cout);
  //   }
  //   else {
  //     for (auto & profiler : operation_profilers_) {
  //       if (profiler->kind() == options_.operation_kind) {
  //         profiler->print_usage(std::cout);
  //         profiler->print_examples(std::cout);
  //         return 0;
  //       }
  //     }
  //   }
  //   return 0;
  // }
  // else if (options_.about.version) {
  //   options_.about.print_version(std::cout);

  //   std::cout << std::endl;
  //   return 0;
  // }
  // else if (options_.about.device_info) {
  //   options_.device.print_device_info(std::cout);
  //   return 0;
  // }

   if (options_.execution_mode == ExecutionMode::kProfile ||
     options_.execution_mode == ExecutionMode::kDryRun ||
     options_.execution_mode == ExecutionMode::kTrace) {

     // Profiles all operations
     return profile_();
   }
   else if (options_.execution_mode == ExecutionMode::kEnumerate) {
      //Enumerates all operations
     enumerate_();
   }
  return 0;
}

/////////////////////////////////////////////////////////////////////////////////////////////////

/// Enumerates all operations
void CutlassProfiler::enumerate_() {

}

/// Profiles all operations
int CutlassProfiler::profile_() {

  int result = 0;
  DeviceContext device_context;
  // For all profilers
  for (auto & profiler : operation_profilers_) {

    if (options_.operation_kind == library::OperationKind::kInvalid ||
      options_.operation_kind == profiler->kind()) {

      result = profiler->profile_all(options_, library::Singleton::get().manifest, device_context);

      if (result) {
        return result;
      }
    }
  }

  return result;
}

/////////////////////////////////////////////////////////////////////////////////////////////////

/// Prints all options
void CutlassProfiler::print_usage_(std::ostream &out) {
}

/// Prints usage
void CutlassProfiler::print_options_(std::ostream &out) {
  options_.print_options(out);
}

/////////////////////////////////////////////////////////////////////////////////////////////////

/// Initializes the CUDA device
void CutlassProfiler::initialize_device_() {

  hipError_t result = hipSetDevice(options_.device.device);

  if (result != hipSuccess) {
    std::cerr << "Failed to set device.";
    throw std::runtime_error("Failed to set device");
  }
}

/////////////////////////////////////////////////////////////////////////////////////////////////

} // namespace profiler
} // namespace cutlass

/////////////////////////////////////////////////////////////////////////////////////////////////
