#include "hip/hip_runtime.h"
/***************************************************************************************************
 * Copyright (c) 2017 - 2024 NVIDIA CORPORATION & AFFILIATES. All rights reserved.
 * SPDX-License-Identifier: BSD-3-Clause
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *
 * 1. Redistributions of source code must retain the above copyright notice, this
 * list of conditions and the following disclaimer.
 *
 * 2. Redistributions in binary form must reproduce the above copyright notice,
 * this list of conditions and the following disclaimer in the documentation
 * and/or other materials provided with the distribution.
 *
 * 3. Neither the name of the copyright holder nor the names of its
 * contributors may be used to endorse or promote products derived from
 * this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
 * AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
 * DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE
 * FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL
 * DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR
 * SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER
 * CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY,
 * OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 *
 **************************************************************************************************/

/*! \file
    \brief CUTLASS Library handle.
*/
#include <iostream>
#include <stdexcept>
#include <cstdint>

#include "cutlass/library/handle.h"
#include "cutlass/library/singleton.h"
#include "cutlass/library/util.h"

namespace cutlass {
namespace library {

///////////////////////////////////////////////////////////////////////////////////////////////////

/// Constructor
Handle::Handle(
  hipStream_t stream,
  size_t workspace_size
):
  provider_(Provider::kCUTLASS),
  stream_(stream),
  workspace_(nullptr),
  workspace_size_(0),
  scalar_pointer_mode_(ScalarPointerMode::kHost),
  last_operation_(nullptr) {

  int device_idx = -1;

  hipError_t error = hipGetDevice(&device_idx);
  if (error != hipSuccess) {
    throw std::runtime_error("hipGetDevice() failed");
  }

  error = hipGetDeviceProperties(&device_, device_idx);
  if (error != hipSuccess) {
    throw std::runtime_error("hipGetDeviceProperties() failed");
  }

  set_workspace_size(workspace_size);

  Singleton::get();
}

/// Destructor
Handle::~Handle() {
  if (workspace_) {

    if (workspace_) {
      hipFree(workspace_);
    }

    workspace_ = nullptr;
    workspace_size_ = 0;
  }
}

/// Move constructor
Handle::Handle(Handle && handle) {
  device_ = handle.device_;
  workspace_size_ = handle.workspace_size_;
  workspace_ = handle.workspace_;
  stream_ = handle.stream_;
  scalar_pointer_mode_ = handle.scalar_pointer_mode_;

  handle.workspace_ = nullptr;
  handle.workspace_size_ = 0;
}

/// Move assignment operator
Handle & Handle::operator=(Handle && handle) {

  provider_ = handle.provider_;
  device_ = handle.device_;
  workspace_size_ = handle.workspace_size_;
  workspace_ = handle.workspace_;
  stream_ = handle.stream_;
  scalar_pointer_mode_ = handle.scalar_pointer_mode_;

  handle.workspace_ = nullptr;
  handle.workspace_size_ = 0;

  return *this;
}

int Handle::compute_capability() const {
  return device_.major * 10 + device_.minor;
}

/// Sets the current CUDA stream
void Handle::set_stream(hipStream_t stream) {
  stream_ = stream;
}

/// Gets the current CUDA stream
hipStream_t Handle::get_stream() const {
  return stream_;
}

/// Gets the current provider
Provider Handle::get_provider() const {
  return provider_;
}

/// Sets the provider of operations
void Handle::set_provider(Provider provider) {
  provider_ = provider;
}

/// Gets the device workspace size
size_t Handle::get_workspace_size() const {
  return workspace_size_;
}

/// Gets a pointer to the device workspace allocation in Global Memory
void *Handle::get_workspace() const {
  return workspace_;
}

/// Sets the size of device workspace, invalidating previous calls to get_device_workspace()
void Handle::set_workspace_size(size_t bytes) {
  if (bytes != workspace_size_) {

    if (workspace_) {
      hipFree(workspace_);
    }

    workspace_ = nullptr;
    workspace_size_ = bytes;

    if (workspace_size_) {

      hipError_t error = hipMalloc((void **)&workspace_, workspace_size_);

      if (error != hipSuccess) {
        throw std::runtime_error("Failed to allocate workspace");
      }
    }
  }

  if (workspace_) {
    hipError_t error = hipMemset(workspace_, 0, workspace_size_);

    if (error != hipSuccess) {
      throw std::runtime_error("Failed to clear workspace");
    }
  }
}

/// Gets the scalar pointer mode
ScalarPointerMode Handle::get_scalar_pointer_mode() const {
  return scalar_pointer_mode_;
}

/// Sets the scalar pointer mode
void Handle::set_scalar_pointer_mode(ScalarPointerMode mode) {
  scalar_pointer_mode_ = mode;
}

/// Gets the last operation
Operation const *Handle::get_last_operation() const {
  return last_operation_;
}

///////////////////////////////////////////////////////////////////////////////////////////////////

/// Returns the maximum required alignment for each operator
static int maximum_alignment_requirement(GemmDescription const &desc) {
  return std::max(
    std::max(desc.A.alignment, desc.B.alignment), desc.C.alignment);
}

/// Returns the largest alignment (in units of elements) the problem satisfies, starting from a
/// given upper limit.
static int gemm_problem_alignment(
  int M,
  int N,
  int K,
  NumericTypeID element_A,
  void const *ptr_A,
  int64_t lda,
  int64_t batch_stride_A,
  NumericTypeID element_B,
  void const *ptr_B,
  int64_t ldb,
  int64_t batch_stride_B,
  NumericTypeID element_C,
  void const * ptr_C,
  int64_t ldc,
  int64_t batch_stride_C,
  void const * ptr_D,
  int64_t ldd,
  int64_t batch_stride_D,
  int max_alignment_in_bytes = 16
) {

  void const *pointers[] = {
    ptr_A, ptr_B, ptr_C, ptr_D
  };

  int64_t extents[] = {
    M, N, K, lda, ldb, ldc, ldd, batch_stride_A, batch_stride_B, batch_stride_C, batch_stride_D
  };

  NumericTypeID elements[] = {
    element_A, element_B, element_C
  };

  for (; max_alignment_in_bytes > 0; max_alignment_in_bytes /= 2) {

    bool satisfied = true;

    // Can pointers satisfy this?
    for (void const *ptr : pointers) {
      std::uintptr_t int_ptr = reinterpret_cast<std::uintptr_t>(ptr);

      if (int_ptr % max_alignment_in_bytes) {
        satisfied = false;
        break;
      }
    }

    if (!satisfied) {
      continue;
    }

    // Compute the maximum alignment based on element data types
    int max_element_alignment = 0;

    for (NumericTypeID type_id : elements) {
      int element_alignment = max_alignment_in_bytes * 8 / library::sizeof_bits(type_id);
      max_element_alignment = std::max(max_element_alignment, element_alignment);
    }

    // Can the problem size and leading dimensions satisfy this?
    for (int64_t extent : extents) {
      if (extent % max_element_alignment) {
        satisfied = false;
        break;
      }
    }

    if (!satisfied) {
      continue;
    }

    // Yes
    return max_element_alignment;
  }

  // No alignment satisfies this problem
  return 0;
}

/// Find the best kernel in descending order of preference.
static Operation const * find_gemm_operation(
  GemmOperationFunctionalMap::const_iterator operators_it,
  GemmPreferenceKey const preference_key) {

  auto cc_it = operators_it->second.upper_bound(preference_key);

  if (cc_it == operators_it->second.begin()) {
    return nullptr;
  }

  Operation const *operation = nullptr;

  // Search in descending order of compute capability
  do {
    --cc_it;

    // Search tile sizes in order, for now.
    for (auto const * op : cc_it->second) {

      GemmDescription const &desc = static_cast<GemmDescription const &>(op->description());

      int min_cc = desc.tile_description.minimum_compute_capability;
      int max_cc = desc.tile_description.maximum_compute_capability;

      int op_alignment = maximum_alignment_requirement(desc);

      if ((min_cc <= preference_key.compute_capability) &&
        (preference_key.compute_capability <= max_cc) &&
        (op_alignment <= preference_key.alignment)) {

        operation = op;
        break;
      }
    }
  } while (!operation && cc_it != operators_it->second.begin());

  return operation;
}

///////////////////////////////////////////////////////////////////////////////////////////////////

/// Executes a GEMM computation: D <= alpha * A*B + beta * C
Status Handle::gemm(

  int M,                                    /// GEMM M dimension
  int N,                                    /// GEMM N dimension
  int K,                                    /// GEMM K dimension

  NumericTypeID element_compute,            /// Data type of internal accumulation

  NumericTypeID element_scalar,             /// Data type of alpha/beta scalars

  void const *alpha,                        /// Pointer to alpha scalar

  NumericTypeID element_A,                  /// Data type of A matrix elements
  LayoutTypeID layout_A,                    /// Layout of A matrix
  ComplexTransform transform_A,             /// Complex transformation applied to A matrix - ignored for real-valued matrices

  void const * ptr_A,                       /// Pointer to A matrix in Global Memory
  int64_t lda,                              /// Leading dimension of A matrix

  NumericTypeID element_B,                  /// Data type of B matrix elements
  LayoutTypeID layout_B,                    /// Layout of B matrix
  ComplexTransform transform_B,             /// Complex transformation applied to B matrix - ignored for real-valued matrices

  void const * ptr_B,                       /// Pointer to B matrix in Global Memory
  int64_t ldb,                              /// Leading dimension of B matrix

  void const * beta,                        /// Pointer to beta scalar

  NumericTypeID element_C,                  /// Data type of C and D matrices

  void const * ptr_C,                       /// Pointer to C matrix
  int64_t ldc,                              /// Leading dimension of C matrix

  void * ptr_D,                             /// Pointer to D matrix
  int64_t ldd                               /// Leading dimension of D matrix
) {

  //
  // Find the operation
  //

  GemmFunctionalKey key(
    provider_,
    GemmKind::kGemm,
    element_compute,
    element_scalar,
    element_A,
    layout_A,
    transform_A,
    element_B,
    layout_B,
    transform_B,
    element_C,  // C/D are same type and col major default
    LayoutTypeID::kColumnMajor,
    element_C,
    LayoutTypeID::kColumnMajor
  );

  auto operators_it = Singleton::get().operation_table.gemm_operations.find(key);

  if (operators_it == Singleton::get().operation_table.gemm_operations.end()) {
    return cutlass::Status::kErrorNotSupported;
  }

  if (operators_it->second.empty()) {
    return cutlass::Status::kErrorNotSupported;
  }

  //
  // Compute the largest alignment restriction the kernel can satisfy.
  //

  // Maximum alignment expectation among all kernels (in units of bytes)
  int const kMaximumAlignmentSize = 16;

  int alignment = gemm_problem_alignment(
    M, N, K,
    element_A, ptr_A, lda, 0,
    element_B, ptr_B, ldb, 0,
    element_C, ptr_C, ldc, 0,
    ptr_D, ldd, 0, kMaximumAlignmentSize
  );

  //
  // Find the best kernel in descending order of preference.
  //

  GemmPreferenceKey preference_key(compute_capability(), alignment);

  Operation const *operation = find_gemm_operation(operators_it, preference_key);

  if (!operation) {
    return cutlass::Status::kErrorNotSupported;
  }

  last_operation_ = operation;

  //
  // Configure operation
  //

  GemmConfiguration configuration{
    {M, N, K},
    lda,
    ldb,
    ldc,
    ldd,
    1
  };

  // Query host work space size
  uint64_t host_workspace_size_needed = operation->get_host_workspace_size(&configuration);

  if (uint64_t(kHostWorkspaceSize) < host_workspace_size_needed) {
    return cutlass::Status::kErrorNotSupported;
  }

  char host_workspace[kHostWorkspaceSize];

  // Query device workspace size
  uint64_t device_workspace_size_needed = operation->get_device_workspace_size(&configuration);

  if (uint64_t(workspace_size_) < device_workspace_size_needed) {
    return cutlass::Status::kErrorNotSupported;
  }

  // Initialize host and device workspaces
  Status status = operation->initialize(
    &configuration,
    host_workspace,
    workspace_,
    stream_);

  if (status != cutlass::Status::kSuccess) {
    return status;
  }

  // Run the operator
  GemmArguments arguments{
    ptr_A,
    ptr_B,
    ptr_C,
    ptr_D,
    alpha,
    beta,
    scalar_pointer_mode_
  };

  return operation->run(&arguments, host_workspace, workspace_, stream_);
}

}
}
